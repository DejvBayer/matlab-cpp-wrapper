#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */

#include <mex/mex.hpp>
#include <mex/Function.hpp>

/*
 * Device code
 */
void __global__ TimesTwo(const double* const A,
                         double* const       B,
                         const unsigned      N)
{
  /* Calculate the global linear index, assuming a 1-d grid. */
  const unsigned i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < N)
  {
    B[i] = 2.0 * A[i];
  }
}

/*
 * Host code
 */
void mex::Function::operator()(Span<Array> lhs, View<ArrayCref> rhs)
{
  /* Declare all variables.*/
  static constexpr char errId[]  = "parallel:gpu:mexGPUExample:InvalidInput";
  static constexpr char errMsg[] = "Invalid input to MEX file.";

  /* Choose a reasonably sized number of threads for the block. */
  static constexpr unsigned threadsPerBlock = 256;

  /* Throw an error if the input is not a GPU array. */
  if (rhs.size() != 1 || !rhs[0].isGpuArray())
  {
    throw mex::Exception{errId, errMsg};
  }

  mex::gpu::Array A{rhs[0]};

  /*
    * Verify that A really is a double array before extracting the pointer.
    */
  if (A.getClassId() != mex::ClassId::_double)
  {
    throw mex::Exception{errId, errMsg};
  }

  /*
    * Now that we have verified the data type, extract a pointer to the input
    * data on the device.
    */
  const double* d_A = mex::gpu::NumericArrayCref<double>{A}.getData();

  /* Create a GPUArray to hold the result and get its underlying pointer. */
  mex::gpu::NumericArray<double> B = mex::gpu::makeUninitNumericArray<double>(A.getDims());

  double* d_B = B.getData();

  /*
    * Call the kernel using the CUDA runtime API. We are using a 1-d grid here,
    * and it would be possible for the number of elements to be too large for
    * the grid. For this example we are not guarding against this possibility.
    */
  const unsigned N = static_cast<unsigned>(A.getSize());
  const unsigned blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  TimesTwo<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);

  /* Wrap the result up as a MATLAB gpuArray for return. */
  lhs[0] = B.release();
}
