#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */

#include <matlabw/mex/mex.hpp>
#include <matlabw/mex/Function.hpp>

using namespace matlabw;

/*
 * Device code
 */
__global__  void TimesTwo(const double* const A,
                          double* const       B,
                          const unsigned      N)
{
  /* Calculate the global linear index, assuming a 1-d grid. */
  const unsigned i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < N)
  {
    B[i] = 2.0 * A[i];
  }
}

/*
 * Host code
 */
void mex::Function::operator()(mx::Span<mx::Array> lhs, mx::View<mx::ArrayCref> rhs)
{
  /* Declare all variables.*/
  static constexpr char errId[]  = "parallel:gpu:mexGPUExample:InvalidInput";
  static constexpr char errMsg[] = "Invalid input to MEX file.";

  /* Choose a reasonably sized number of threads for the block. */
  static constexpr unsigned threadsPerBlock = 256;

  /* Throw an error if the input is not a GPU array. */
  if (rhs.size() != 1 || !rhs[0].isGpuArray())
  {
    throw mx::Exception{errId, errMsg};
  }

  mx::gpu::Array A{rhs[0]};

  /*
    * Verify that A really is a double array before extracting the pointer.
    */
  if (A.getClassId() != mx::ClassId::_double)
  {
    throw mx::Exception{errId, errMsg};
  }

  /*
    * Now that we have verified the data type, extract a pointer to the input
    * data on the device.
    */
  const double* d_A = mx::gpu::NumericArrayCref<double>{A}.getData();

  /* Create a GPUArray to hold the result and get its underlying pointer. */
  auto B = mx::gpu::makeUninitNumericArray<double>(A.getDims());

  double* d_B = B.getData();

  /*
    * Call the kernel using the CUDA runtime API. We are using a 1-d grid here,
    * and it would be possible for the number of elements to be too large for
    * the grid. For this example we are not guarding against this possibility.
    */
  const unsigned N = static_cast<unsigned>(A.getSize());
  const unsigned blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
  TimesTwo<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, N);

  /* Wrap the result up as a MATLAB gpuArray for return. */
  lhs[0] = B.release();
}
